#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/huber_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void HuberForward(const int n, Dtype* loss, Dtype* diff,
    const Dtype* pre_data, const Dtype* gt_data, const Dtype delta) {
  Dtype delta_2 = delta * delta;
  CUDA_KERNEL_LOOP(index, n) {
    Dtype abs_diff = abs(diff[index]);
    if (abs_diff < delta) {
      loss[index] = (0.5 * abs_diff * abs_diff);
      diff[index] = -diff[index];
    } else {
      loss[index] = (delta * abs_diff - 0.5*delta_2);
      diff[index] = (pre_data[index] < gt_data[index] ? -delta : delta);
    }
  }
}

template <typename Dtype>
void HuberLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // bottom[0] is predicted values
  // bottom[1] is ground truth
  int count = bottom[0]->count();
  const Dtype* pre_data = bottom[0]->gpu_data();
  const Dtype* gt_data = bottom[1]->gpu_data();
  // For accelerating backward compute,
  // compute gradient in forward (store in diff_)
  Dtype* diff_data = diff_.mutable_gpu_data();

  caffe_gpu_sub(count, gt_data, pre_data, diff_data);

  const Dtype delta = this->layer_param_.huber_loss_param().delta();

  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();

  // NOLINT_NEXT_LINE(whitespace/operators)
  HuberForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, loss_data, diff_data, pre_data, gt_data, delta);
  CUDA_POST_KERNEL_CHECK;

  Dtype loss;
  caffe_gpu_asum(count, loss_data, &loss);
  top[0]->mutable_cpu_data()[0] = (loss / count);
}

template <typename Dtype>
void HuberLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    int count = bottom[0]->count();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, diff_.gpu_data(), bottom_diff);
    // Scale gradient
    Dtype loss_weight = top[0]->cpu_diff()[0] / count;
    caffe_gpu_scal(count, loss_weight, bottom_diff);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(HuberLossLayer);
}  // namespace caffe
